﻿#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void square(float* d_out, float* d_in)
{
	int idx = threadIdx.x;
	float f = d_in[idx];
	d_out[idx] = f * f;
}
int main(int argc, char** argv)
{
	const int ARRAY_SIZE = 1024;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);
	
	// generate the input array on the host 
	float h_in[ARRAY_SIZE];
	for (int i = 0; i <ARRAY_SIZE; i++)
		h_in[i] = float(i);
	float h_out[ARRAY_SIZE];
	
	// generate GPU memory pointers
	float* d_in;
	float* d_out;
	
	// allocate GPU memory
	hipMalloc((void**)&d_in, ARRAY_BYTES);
	hipMalloc((void**)&d_out, ARRAY_BYTES);
	
	//transfer the array to the GPU
	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);
	
	//launch kernel
	square<<<2, ARRAY_SIZE >>>(d_out, d_in);
	
	// copy back the result array to the CPU
	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);
	
	//print out the result array to the CPU
	for (int i = 0; i < ARRAY_SIZE; i++)
	{
		printf("%f", h_out[i]);
		printf(((i % 4) != 3) ? "\t" : "\n");
	}
	
	hipFree(d_in);
	hipFree(d_out);
	return 0;
}
